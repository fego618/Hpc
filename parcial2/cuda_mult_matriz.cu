#include <hip/hip_runtime.h>
#include <cstdio>
#define BLOCKSIZE 32
__global__ void kernelMultMat(double *d_a, double *d_b, double *d_c, int ROWS, int COLA, int COLB) {
  
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  double add;

  if (row < ROWS && col < COLB) {
    add = 0;
    for (int k = 0; k < COLA; k++) {
      add += d_a[row * COLA + k] * d_b[k * COLB + col];
    }
    d_c[row * COLB + col] = add;
  }
}


void cuda_mult_matriz(double *h_a, double *h_b, double *h_c, int ROWS, int COLA, int COLB){
	
	double *d_a,*d_b,*d_c;

	int sizeA = ROWS*COLA;
	int sizeB = COLA*COLB;
	int sizeC = ROWS*COLB;

	hipMalloc(&d_a, sizeof(double)*sizeA);
	hipMalloc(&d_b, sizeof(double)*sizeB);
	hipMalloc(&d_c, sizeof(double)*sizeC);

	hipMemcpy(d_a,h_a,sizeA * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,sizeB * sizeof(double), hipMemcpyHostToDevice);

	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);
	dim3 dimGrid((COLB / BLOCKSIZE) + 1, (ROWS / BLOCKSIZE) + 1,1);

	kernelMultMat<<< dimGrid, dimBlock >>>(d_a, d_b, d_c, ROWS, COLA, COLB);
	hipMemcpy(h_c, d_c, sizeC*sizeof(double),hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

}